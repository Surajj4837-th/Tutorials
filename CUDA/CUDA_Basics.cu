#include "hip/hip_runtime.h"
// CUDA basics
#include <stdio.h>
#include "Common/book.h"


__global__ void kernel(void)
{
    printf("Hello World!!!\n");
}


int main()
{
    kernel <<<1,1>>> ();

    hipDeviceProp_t  prop;
    int dev;

    //Get current CUDA device properties.
    HANDLE_ERROR( hipGetDevice( &dev ) );
    printf( "ID of current CUDA device:  %d\n", dev );

    //get CUDA device which is closest to revision 1.3
    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    //Set CUDA device
    HANDLE_ERROR( hipSetDevice( dev ) );

    return 0;

}
/************************************Summary*****************************************
1. Cuda files are to be save with .cu extension.

2. __global__ qualifier informs compiler that the function should be compiled to run on 
   device instead of host.

3. 
************************************************************************************/