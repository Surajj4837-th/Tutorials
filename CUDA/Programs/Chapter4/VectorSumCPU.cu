#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "../../Common/book.h"
#include <time.h>

#define N 100000000 

void add(int *a, int *b, int *c)
{
    int tid = 0;
    // this is CPU zero, so we start at zero
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += 1;
        // we have one CPU, so we increment by one
    }
}

int main(void)
{
    int *a, *b, *c;
    clock_t start, end;
    double cpu_time_used;    

    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * 2;
    }

    start = clock();
    add(a, b, c);
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("add() took %f seconds to execute \n", cpu_time_used); 
    
    // display the results
    // for (int i = 0; i < N; i++)
    // {
    //     printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }
    
    return 0;
}