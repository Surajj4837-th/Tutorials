#include "hip/hip_runtime.h"
// Memory allocation
#include <stdio.h>
#include <sys/time.h>
#include "Dep/common/book.h"

#define N   10

__global__ void add( int *a, int *b, int *c ) 
{
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}


void add_CPU( int *a, int *b, int *c ) {
    int tid = 0;    // this is CPU zero, so we start at zero
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += 1;   // we have one CPU, so we increment by one
    }
}


int main( void ) 
{
    int a[N], b[N], c[N];       //local variables
    int *dev_a, *dev_b, *dev_c; //device pointers

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) 
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    struct timeval t1, t2;
    gettimeofday(&t1, 0);
    add<<<N,1>>>( dev_a, dev_b, dev_c );
    //add_CPU( a, b, c );
    gettimeofday(&t2, 0);

    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to generate:  %3.1f us \n", time);


    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

    // display the results
    for (int i=0; i<N; i++) 
    {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}
/************************************Summary*****************************************
1. Cuda files are to be save with .cu extension.

2. __global__ qualifier informs compiler that the function should be compiled to run on 
   device instead of host.

3. 
************************************************************************************/